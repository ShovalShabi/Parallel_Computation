#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"


__device__ double calculateDistance(const Point p1,const Point p2, double t){
    double xP1, yP1, xP2, yP2;

    xP1 = ((p1.x2 - p1.x1) / 2 ) * sin (t*M_PI /2) + (p1.x2 + p1.x1) / 2; 
    yP1 = p1.a*xP1 + p1.b;

    xP2 = ((p2.x2 - p2.x1) / 2 ) * sin (t*M_PI /2) + (p2.x2 + p2.x1) / 2; 
    yP2 = p2.a*xP2 + p2.b;


    return sqrt(pow(xP2-xP1,2) + pow(yP2-yP1,2));
}


__global__ void findProximityCriteria(Point* pointsArrDevice, int nCount, double* actualTsDevice, int** tidAndPidsDevice, int tCount, int proximity, double distance, int minTIndex, int maxTIndex) {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadId < nCount * tCount){
        int indexPoint = threadId % nCount;  // The index of the point within the buffer
        int indexT = threadId / nCount;  // The index of the current t value

        //Making sure that the process caclute only the range of t values that assigned to it
        if ( minTIndex <= indexT <= maxTIndex  ){
            int count = 0;
            for (int i = 0; i < nCount && i != indexPoint; i++){
                double dist = calculateDistance(pointsArrDevice[indexPoint], pointsArrDevice[i], actualTsDevice[indexT]);

                if (dist <= distance)
                    count++;
                
                if (count == proximity)
                    break;  
            }

            if (count == proximity){
                for (int j = 0; j < CONSTRAINT; j++)
                    if(!tidAndPidsDevice[indexT][j])
                        atomicExch(&tidAndPidsDevice[indexT][j],pointsArrDevice[indexPoint].id);
            }
        }
    }
}

__global__ void intializeTidsAndPids(int* tidsAndPidsDevice){
    int threadId = threadIdx.x;

    tidsAndPidsDevice[threadId] = -1;
}



int computeOnGPU(Point* pointArr, int numPoints, double* actualTs, int** tidsAndPids , int tCount, int proximity, double distance, int minTIndex, int maxTIndex) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    size_t pitch;

    // Allocate memory on device for overall points buffer on device
    Point* pointsArrDevice = NULL;
    err = hipMalloc((void**)&pointsArrDevice, numPoints * sizeof(Point));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copying memory on device for overall points buffer on device
    err = hipMemcpy(pointsArrDevice, pointArr, numPoints * sizeof(Point), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate memory on device for actual t values buffer on device
    double* actualTsDevice = NULL;
    err = hipMalloc((void**) &actualTsDevice, tCount * sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // Copying memory on device for actual t values buffer on device
    err = hipMemcpy(actualTsDevice, actualTs, tCount * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the matching tids and pids to two dimensional array to the device
    int** tidsAndPidsDevice = NULL;
    err = hipMallocPitch((void**) tidsAndPidsDevice, &pitch, tCount * sizeof(int), CONSTRAINT);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copying the matching tids and pids to two dimensional array to the device
    err = hipMemcpy2D(tidsAndPidsDevice, pitch, tidsAndPids, tCount * sizeof(int), tCount * sizeof(int) , CONSTRAINT, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    intializeTidsAndPids <<<1, tCount * CONSTRAINT>>>((int*) tidsAndPids);

    for (int i = 0; i < tCount; i++)
    {
        for (int j = 0; j < CONSTRAINT; j++)
        {
            printf("tidsAndPidsDevice[%d][%d] = %d",i,j,tidsAndPidsDevice[i][j]);
        }
        
    }
    

    int numBlocks = (int) ceil( numPoints * tCount / THREADS_PER_BLOCK);

    // Finding all the Proximity Criteria of each distinct t value
    findProximityCriteria<<<numBlocks, THREADS_PER_BLOCK>>>(pointsArrDevice, numPoints, actualTsDevice, tidsAndPidsDevice, tCount, proximity, distance, minTIndex, maxTIndex);

    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Synchronize to ensure all CUDA operations are completed
    hipDeviceSynchronize();

    // Copy the final histogram from the device to the host
    err = hipMemcpy2D(tidsAndPids, tCount * sizeof(int), tidsAndPidsDevice, pitch, tCount * sizeof(int), CONSTRAINT, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free device global memory
    err = hipFree(pointsArrDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free device global memory
    err = hipFree(actualTsDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free device global memory
    err = hipFree(tidsAndPidsDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}