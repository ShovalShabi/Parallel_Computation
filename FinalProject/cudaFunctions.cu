#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"

/**
 * @brief Calculate the distance between two points.
 * @param p1 First point.
 * @param p2 Second point.
 * @param t T value.
 * @return Distance between the two points.
 */
__device__ double calculateDistance(const Point p1,const Point p2, double t){
    double xP1, yP1, xP2, yP2;

    xP1 = ((p1.x2 - p1.x1) / 2 ) * sin (t*M_PI /2) + (p1.x2 + p1.x1) / 2; 
    yP1 = p1.a*xP1 + p1.b;

    xP2 = ((p2.x2 - p2.x1) / 2 ) * sin (t*M_PI /2) + (p2.x2 + p2.x1) / 2; 
    yP2 = p2.a*xP2 + p2.b;

    return sqrt(pow(xP2-xP1,2) + pow(yP2-yP1,2));
}

/**
 * @brief Find the proximity criteria for each point and t value on the GPU.
 * @param pointsArrDevice Device array of points.
 * @param nCount Total number of points.
 * @param actualTsDevice Device array of actual t values.
 * @param tidAndPidsDevice Device array of tids and pids.
 * @param tCount Total number of t values.
 * @param proximity Proximity value.
 * @param distance Distance value.
 * @param minTIndex Minimum t index.
 * @param maxTIndex Maximum t index.
 */
__global__ void findProximityCriteria(Point* pointsArrDevice, int nCount, double* actualTsDevice, int* tidAndPidsDevice, int tCount, int proximity, double distance, int minTIndex, int maxTIndex) {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadId < nCount * tCount){
        int indexPoint = threadId % nCount;  // The index of the point within the buffer
        int indexT = threadId / nCount;  // The index of the current t value

        //Making sure that the process calculates only the range of t values that are assigned to it
        if (minTIndex <= indexT <= maxTIndex){
            int count = 0;
            for (int i = 0; i < nCount && i != indexPoint; i++){
                double dist = calculateDistance(pointsArrDevice[indexPoint], pointsArrDevice[i], actualTsDevice[indexT]);

                if (dist <= distance)
                    count++;
                
                if (count == proximity)
                    break;  
            }

            //The exmined point reached K neighbors which is proximity values
            if (count == proximity){
                for (int j = 0; j < CONSTRAINT; j++){
                    if (tidAndPidsDevice[indexT * CONSTRAINT + j] == pointsArrDevice[indexPoint].id)
                        break;
                    
                    if(tidAndPidsDevice[indexT * CONSTRAINT + j] < 0){
                        atomicExch(&tidAndPidsDevice[indexT * CONSTRAINT + j],pointsArrDevice[indexPoint].id);
                        break;
                    }    
                }
            }
        }
    }
}

/**
 * @brief Initialize the tids and pids array on the GPU, -1 means that the slot is free to assignment.
 * @param tidsAndPidsDevice Device array of tids and pids.
 */
__global__ void intializeTidsAndPids(int* tidsAndPidsDevice){
    int threadId = threadIdx.x;

    tidsAndPidsDevice[threadId] = -1;
}

/**
 * @brief Perform the GPU computation for finding proximity criteria.
 * @param pointArr Array of points.
 * @param numPoints Number of points.
 * @param actualTs Array of actual t values.
 * @param tidsAndPids 2D array of tids and pids.
 * @param tCount Number of t values.
 * @param proximity Proximity value.
 * @param distance Distance value.
 * @param minTIndex Minimum t index.
 * @param maxTIndex Maximum t index.
 * @return 0 on success.
 */
int computeOnGPU(Point* pointArr, int numPoints, double* actualTs, int** tidsAndPids , int tCount, int proximity, double distance, int minTIndex, int maxTIndex) {

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Allocate memory on device for overall points buffer on device
    Point* pointsArrDevice = NULL;
    err = hipMalloc((void**)&pointsArrDevice, numPoints * sizeof(Point));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy memory to device for overall points buffer on device
    err = hipMemcpy(pointsArrDevice, pointArr, numPoints * sizeof(Point), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate memory on device for actual t values buffer on device
    double* actualTsDevice = NULL;
    err = hipMalloc((void**)&actualTsDevice, tCount * sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // Copy memory to device for actual t values buffer on device
    err = hipMemcpy(actualTsDevice, actualTs, tCount * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate memory for the 2D array on the device
    int* tidsAndPidsDevice = NULL;
    err = hipMalloc((void**)&tidsAndPidsDevice, tCount * CONSTRAINT * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    intializeTidsAndPids <<<1, tCount * CONSTRAINT>>>(tidsAndPidsDevice);

    
    //Calculation of the efficient number of blocks to CUDA threads (256 threads per block)
    int numBlocks = (int) ceil( numPoints * tCount / THREADS_PER_BLOCK);

    // Finding all the Proximity Criteria of each distinct t value
    findProximityCriteria<<<numBlocks, THREADS_PER_BLOCK>>>(pointsArrDevice, numPoints, actualTsDevice, tidsAndPidsDevice, tCount, proximity, distance, minTIndex, maxTIndex);

    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Synchronize to ensure all CUDA operations are completed
    hipDeviceSynchronize();

    // Copy the final tid and pid array from the device to the host
    for (int i = minTIndex; i <= maxTIndex; i++){
        
        //Copying under each t index the ids of the Proximity Criteria points
        err =  hipMemcpy(tidsAndPids[i],tidsAndPidsDevice + i * CONSTRAINT, CONSTRAINT * sizeof(int),hipMemcpyDeviceToHost);

        if (err != hipSuccess) {
            fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }

    // Free device global memory
    err = hipFree(pointsArrDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free device global memory
    err = hipFree(actualTsDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free device global memory
    err = hipFree(tidsAndPidsDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Reset the device
    err = hipDeviceReset();
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return 0;
}
