#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"


__device__ double calculateDistance(const Point p1,const Point p2, double t){
    double xP1, yP1, xP2, yP2;

    xP1 = ((p1.x2 - p1.x1) / 2 ) * sin (t*M_PI /2) + (p1.x2 + p1.x1) / 2; 
    yP1 = p1.a*xP1 + p1.b;

    xP2 = ((p2.x2 - p2.x1) / 2 ) * sin (t*M_PI /2) + (p2.x2 + p2.x1) / 2; 
    yP2 = p2.a*xP2 + p2.b;


    return sqrt(pow(xP2-xP1,2) + pow(yP2-yP1,2));
}


__global__ void findProximityCriteria(Point* pointsArrDevice, int nCount, double* actualTsDevice, int* tidAndPidsDevice, int tCount, int proximity, double distance, int minTIndex, int maxTIndex) {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadId < nCount * tCount){
        int indexPoint = threadId % nCount;  // The index of the point within the buffer
        int indexT = threadId / nCount;  // The index of the current t value

        //printf("Thread %d calculating point %d and t[%d]=%lf\n",threadId,pointsArrDevice[indexPoint].id,indexT,actualTsDevice[indexT]);

        //Making sure that the process caclute only the range of t values that assigned to it
        if ( minTIndex <= indexT <= maxTIndex  ){
            int count = 0;
            for (int i = 0; i < nCount && i != indexPoint; i++){
                double dist = calculateDistance(pointsArrDevice[indexPoint], pointsArrDevice[i], actualTsDevice[indexT]);

                if (dist <= distance)
                    count++;
                
                if (count == proximity)
                    break;  
            }

            if (count == proximity){
                for (int j = 0; j < CONSTRAINT; j++){
                    if (tidAndPidsDevice[indexT * CONSTRAINT + j] == pointsArrDevice[indexPoint].id)
                        break;
                    
                    if(tidAndPidsDevice[indexT * CONSTRAINT + j] < 0){
                        atomicExch(&tidAndPidsDevice[indexT * CONSTRAINT + j],pointsArrDevice[indexPoint].id);
                        // printf("Thread %d found point %d as proximity point at t[%d]=%lf\n",threadId,pointsArrDevice[indexPoint].id,indexT,actualTsDevice[indexT]);
                        break;
                    }    
                }
            }
        }
    }
}

__global__ void intializeTidsAndPids(int* tidsAndPidsDevice){
    int threadId = threadIdx.x;

    tidsAndPidsDevice[threadId] = -1;
}



int computeOnGPU(Point* pointArr, int numPoints, double* actualTs, int** tidsAndPids , int tCount, int proximity, double distance, int minTIndex, int maxTIndex) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    // size_t pitch;

    // Allocate memory on device for overall points buffer on device
    Point* pointsArrDevice = NULL;
    err = hipMalloc((void**)&pointsArrDevice, numPoints * sizeof(Point));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copying memory on device for overall points buffer on device
    err = hipMemcpy(pointsArrDevice, pointArr, numPoints * sizeof(Point), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate memory on device for actual t values buffer on device
    double* actualTsDevice = NULL;
    err = hipMalloc((void**)&actualTsDevice, tCount * sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // Copying memory on device for actual t values buffer on device
    err = hipMemcpy(actualTsDevice, actualTs, tCount * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate memory for whole columns of the 2D array on the device
    int* tidsAndPidsDevice = NULL;
    err = hipMalloc((void**)&tidsAndPidsDevice, tCount * CONSTRAINT * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    intializeTidsAndPids <<<1, tCount * CONSTRAINT>>>(tidsAndPidsDevice);

    printf("Initilized tidsAndPids\n");
    
    int numBlocks = (int) ceil( numPoints * tCount / THREADS_PER_BLOCK);

    // Finding all the Proximity Criteria of each distinct t value
    findProximityCriteria<<<numBlocks, THREADS_PER_BLOCK>>>(pointsArrDevice, numPoints, actualTsDevice, tidsAndPidsDevice, tCount, proximity, distance, minTIndex, maxTIndex);

    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Synchronize to ensure all CUDA operations are completed
    hipDeviceSynchronize();

    printf("Finished to find ProximtyCriteria\n");

    // Copy the final histogram from the device to the host
    for (int i = minTIndex; i <= maxTIndex; i++){
        //Copying under each t index the ids of the ProximityCriteria points
        printf("Copying values to tidsAndPids[%d]\n",i);
        err =  hipMemcpy(tidsAndPids[i],tidsAndPidsDevice + i * CONSTRAINT, CONSTRAINT * sizeof(int),hipMemcpyDeviceToHost);

        if (err != hipSuccess) {
            fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }

    printf("Copied all tidsAndPids\n");

    for (int i = minTIndex; i <= maxTIndex; i++)
    {
        for (int j = 0; j < CONSTRAINT; j++)
        {
            printf("The process in ranges %d -- %d seen tidsAndPids[%d][%d] = %d\n",minTIndex,maxTIndex,i,j,tidsAndPids[i][j]);
        }
        
    }
     
    // Free device global memory
    err = hipFree(pointsArrDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free device global memory
    err = hipFree(actualTsDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free device global memory
    err = hipFree(tidsAndPidsDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}