#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"



  __global__  void buildHist(int* data, int dataSize, int* hist) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int chunck = (dataSize/(NUM_BLOCKS*NUM_THREADS));
    for (int i = index*chunck ; i < index*chunck+chunck; i++)
        atomicAdd(&hist[data[i]],1);
    
  }


__global__  void initHist(int * h) {

  int index = threadIdx.x;
  h[index] = 0;

}

int computeOnGPU(int *data, int dataSize ,int* histValues) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    size_t size = dataSize * sizeof(int);

    // Allocate memory on device
    int *device_data = NULL;
    err = hipMalloc((void **)&device_data, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int *device_hist = NULL;
    err = hipMalloc((void **)&device_hist, RANGE * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Copy the vector A to the device
    err = hipMemcpy(device_data, data, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Initialize vectors on device, 1 block and number of threads AS  RANGE
    initHist <<< 1 , RANGE >>> (device_hist);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Unify the results
    buildHist<<<  NUM_BLOCKS,  NUM_THREADS>>>(device_data,dataSize, device_hist);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();

    // Copy the final histogram to the host
    err = hipMemcpy(histValues,device_hist,sizeof(int)*RANGE,hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free device global memory
    err = hipFree(device_data);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(device_hist);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}

